#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "cudaFuncs.h"

__global__ void calc_ij_kernel(const int picN, const int objN, const int* picMat, const int* objMat, double* resMat) {
	for(int i = threadIdx.x; i < picN; i += MAX_MAT_WIDTH) {
		for(int j = threadIdx.y; j < picN; j += MAX_MAT_WIDTH) {
	
			if(i+objN > picN || j+objN > picN)
				resMat[i*picN + j] = -1;	
			else {
				double sum = 0;
				for(int k = 0; k < objN; k++)
					for(int t = 0; t < objN; t++)
						sum += fabs((double)(picMat[(i+k)*picN + j+t] - objMat[k*objN + t]) / (double)picMat[(i+k)*picN + j+t]);
					
				resMat[i*picN + j] = sum;
			}
		}
	}
}

void handleError(hipError_t err, int msgChoice) {
    	if (err != hipSuccess) {
	    	if(msgChoice == 1)
        		fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        	if(msgChoice == 2)
        		fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	if(msgChoice == 3)
	        	fprintf(stderr, "Failed to launch kernel -  %s\n", hipGetErrorString(err));
        	if(msgChoice == 4)
        		fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        	else
        		fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
}

void cudaFindMatch(const int match_val, const int numObjects, struct pic_obj pic, struct pic_obj *Objects, int **result) {
	int gridSize, flag = 0;
	(*result)[0] = pic.id; (*result)[1] = -1; (*result)[2] = -1; (*result)[3] = -1;
	
    	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
  
    	// Allocate memory on GPU to copy the data from the host
    	int *d_picMat;
    	//double *d_matRes, *matRes;
    	    	
    	err = hipMalloc((void **)&d_picMat, pic.N*pic.N * sizeof(int));
    	handleError(err, 1);
    	    	
    	////////////////////////////////////////////////////////
    	
    	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_picMat, pic.mat, pic.N*pic.N * sizeof(int), hipMemcpyHostToDevice);
    	handleError(err, 2);
    	
    	for(int obj = 0; obj < numObjects; obj++) {
    		int *d_objMat;
	    	double *d_matRes, *matRes;
	    	
    		err = hipMalloc((void **)&d_objMat, Objects[obj].N*Objects[obj].N * sizeof(int));
    		handleError(err, 1);
    		
    		err = hipMalloc((void **)&d_matRes, pic.N*pic.N * sizeof(double));
	    	handleError(err, 1);
    	
	    	err = hipMemcpy(d_objMat, Objects[obj].mat, Objects[obj].N*Objects[obj].N * sizeof(int), hipMemcpyHostToDevice);
	    	handleError(err, 2);

	    	// Launch the Kernel
	    	if(pic.N <= MAX_MAT_WIDTH) {
	    		dim3 dimBlock(pic.N, pic.N);
		    	calc_ij_kernel<<<1, dimBlock>>>(pic.N, Objects[obj].N, d_picMat, d_objMat, d_matRes);
		    	handleError(hipGetLastError(), 3);
		}
		else {
		    	//gridSize = (pic.N + MAX_MAT_WIDTH - 1) / MAX_MAT_WIDTH;
		    	gridSize = pic.N % MAX_MAT_WIDTH ? pic.N / MAX_MAT_WIDTH + 1 : pic.N / MAX_MAT_WIDTH;
		    	dim3 dimGrid(gridSize, gridSize);
		    	dim3 dimBlock(MAX_MAT_WIDTH, MAX_MAT_WIDTH);
		    	calc_ij_kernel<<<1, dimBlock>>>(pic.N, Objects[obj].N, d_picMat, d_objMat, d_matRes);
		    	handleError(hipGetLastError(), 3);
	    	}
	    	
	    	matRes = (double*)malloc(pic.N*pic.N * sizeof(double));
	    	if (matRes == NULL) {
			printf("Problem to allocate memory\n");
			exit(0);
		}
    		err = hipMemcpy(matRes, d_matRes, pic.N*pic.N * sizeof(double), hipMemcpyDeviceToHost);
		handleError(err, 4);
		
		for(int i = 0; i < pic.N && !flag; i++) {
			for(int j = 0; j < pic.N && !flag; j++) {
				if(matRes[i*pic.N + j] <= match_val && matRes[i*pic.N + j] >= 0) {
					(*result)[1] = Objects[obj].id; (*result)[2] = i; (*result)[3] = j;
					flag = 1;
				}
			}
		}
	    	
	        err = hipFree(d_objMat);
	        handleError(err, 5);
	        
	        err = hipFree(d_matRes);
	        handleError(err, 5);
	        
	        free(matRes);
	        
	        if(flag)
	        	break;
	}	 
	
    	// Free allocated memory on GPU
    	err = hipFree(d_picMat);
    	handleError(err, 5);
}
